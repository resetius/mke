
#include <hip/hip_runtime.h>
/* -*- charset: utf-8 -*- */
/*$Id: linal_base.cpp,v b772905ca13d 2009/08/07 07:39:47 aozeritsky $*/

/* Copyright (c) 2009 Alexey Ozeritsky (������� ���������)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. Redistributions in any form must be accompanied by information on
 *    how to obtain complete source code for the Phelm software and any
 *    accompanying software that uses the Phelm software.  The source code
 *    must either be included in the distribution or be available for no
 *    more than the cost of distribution plus a nominal fee, and must be
 *    freely redistributable under reasonable conditions.  For an
 *    executable file, complete source code means the source code for all
 *    modules it contains.  It does not include source code for modules or
 *    files that typically accompany the major components of the operating
 *    system on which the executable file runs.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR ``AS IS'' AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES
 * OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED.
 * IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT
 * NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF
 * THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

__global__ void _sparse_mult_vector_ld(double * r, 
	const int * Ap, 
	const int * Ai, 
	const double * Ax,
	const double * x, 
	int n)
{
}

void sparse_mult_vector_ld(double * r, 
	const int * Ap, 
	const int * Ai, 
	const double * Ax,
	const double * x, 
	int n)
{
	_sparse_mult_vector_ld <<< 4, 4 >>> (r, Ap, Ai, Ax, x, n);
}

__global__ void _sparse_mult_vector_lf(float * r, 
	const int * Ap, 
	const int * Ai, 
	const float * Ax,
	const float * x, 
	int n)
{
}

void sparse_mult_vector_lf(float * r, 
	const int * Ap, 
	const int * Ai, 
	const float * Ax,
	const float * x, 
	int n)
{
	_sparse_mult_vector_lf <<< 4, 4 >>> (r, Ap, Ai, Ax, x, n);
}
